#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "SphereMath.h"
#include "RayTracing.h"
#include "SphereCreator.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float t, Sphere* ptrDevTabSphere);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float t, Sphere* ptrDevTabSphere)
    {

    //math
//    size_t sizeOctet = nbSphere * sizeof(Sphere);
//    Device::memcpyHToD(ptrTabSphere, ptrDevTabSphere, sizeOctet);
    SphereMath sphereMath = SphereMath(w, h, nbSphere, ptrDevTabSphere); // ici pour preparer cuda

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;
    int j;

    int s = TID; // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

//	domainMath.toXY(i, j, &x, &y);
	sphereMath.colorIJ(&ptrDevPixels[s], i, j, t);

	s += NB_THREAD;
	}

    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

//    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
