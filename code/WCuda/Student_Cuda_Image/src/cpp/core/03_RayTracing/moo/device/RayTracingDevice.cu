#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "SphereMath.h"
#include "RayTracing.h"
#include "SphereCreator.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include "length_cm.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

__constant__ Sphere TAB_CM[LENGTH_CM];

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
//__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float t, Sphere* ptrDevTabSphere);
//__global__ void rayTracingGM(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float dt, Sphere* ptrDevTabSphere);
__global__ void rayTracingSM(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float dt, Sphere* ptrDevTabSphere);
__device__ void copyGMtoSM(Sphere* ptrDevTabSphere, Sphere* tab_SM);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/
__host__ void uploadGPU(Sphere* tabValue)
    {
    size_t size = LENGTH_CM * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), tabValue, size, offset, hipMemcpyHostToDevice));
    }

__device__ void work(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float t, Sphere* ptrDevTabSphere)
    {

    SphereMath sphereMath = SphereMath(w, h, nbSphere, ptrDevTabSphere); // ici pour preparer cuda

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int i;
    int j;

    int s = TID; // in [0,...
    while (s < WH)
	{
	// the algorithme
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

	//	domainMath.toXY(i, j, &x, &y);
	sphereMath.colorIJ(&ptrDevPixels[s], i, j, t);

	s += NB_THREAD;
	}
    }

__device__ void copyGMtoSM(Sphere* ptrDevTabSphere, Sphere* tab_SM)
    {
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();

    int s = TID_LOCAL;

    //pour la comprehenssion...
    int NB_SPHERE = LENGTH_CM;

    while (s < NB_SPHERE)
	{
	tab_SM[s] = ptrDevTabSphere[s];
	s += NB_THREAD_LOCAL;
	}
    }

__global__ void rayTracingCM(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    work(ptrDevPixels, w, h, LENGTH_CM, t, TAB_CM);
    }
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
//---rayTracingGM
__global__ void rayTracingGM(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float t, Sphere* ptrDevTabSphere)
    {
    work(ptrDevPixels, w, h, nbSphere, t, ptrDevTabSphere);
    }

__global__ void rayTracingSM(uchar4* ptrDevPixels, uint w, uint h, int nbSphere, float t, Sphere* ptrDevTabSphere)
    {
    extern __shared__ Sphere tab_SM[];
    copyGMtoSM(ptrDevTabSphere, tab_SM);

    __syncthreads();
    work(ptrDevPixels, w, h, nbSphere, t, tab_SM);
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

//    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
