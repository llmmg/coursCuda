#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "RayTracing.h"
#include "SphereCreator.h"
#include "SphereMath.h"
#include <assert.h>

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"

using namespace gpu;
using std::cout;
using std::endl;
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
extern __global__ void rayTracing(uchar4* ptrDevPixels,uint w, uint h, int nbSphere,float dt, Sphere* ptrDevTabSphere);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
RayTracing::RayTracing(const Grid& grid, uint w, uint h, int nbSphere, float dt/*, Sphere* ptrTabSphere*/) :
	Animable_I<uchar4>(grid, w, h, "RayTracig_CUDA_RGBA_uchar4")
    {

    this->nbSphere = nbSphere;
//    this->ptrTabSphere = new Sphere[nbSphere];
//    this->w = w;
//    this->h = h;
    this->dt = dt; // protected dans Animable

    SphereCreator sphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    // toGM(ptrTabSphere);
    this->sizeOctet = nbSphere * sizeof(Sphere);

    Device::malloc(&ptrDevTabSphere, sizeOctet);
    Device::memclear(ptrDevTabSphere, sizeOctet);

    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);

    //  toCM(ptrTabSphere);
    //???

    //TODO: verifier les pointeurs et sense des allocation
    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
//    Device::free(ptrTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h/*, int nbSphere, Sphere* ptrTabSphere*/, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("fractale rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    rayTracing<<<dg,db>>>(ptrDevPixels, w, h, nbSphere, dt, ptrDevTabSphere);
    // le kernel est importer ci-dessus (ligne 19)

    Device::lastCudaError("fractale rgba uchar4 (after)"); // facultatif, for debug only, remove for release

//    Device::memcpyDToH(ptrTabSphere, ptrDevTabSphere, sizeOctet);
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
//    n = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

