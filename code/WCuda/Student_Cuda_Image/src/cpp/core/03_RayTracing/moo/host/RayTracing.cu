#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "RayTracing.h"
#include "SphereCreator.h"
#include "SphereMath.h"
#include <assert.h>

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"

#include "length_cm.h"

using namespace gpu;
using std::cout;
using std::endl;
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
//extern __global__ void rayTracing(uchar4* ptrDevPixels,uint w, uint h, int nbSphere,float dt, Sphere* ptrDevTabSphere);
extern __global__ void rayTracingGM(uchar4* ptrDevPixels,uint w, uint h, int nbSphere,float dt, Sphere* ptrDevTabSphere);
extern __global__ void rayTracingCM(uchar4* ptrDevPixels, uint w, uint h, float t);
extern __global__ void rayTracingSM(uchar4* ptrDevPixels,uint w, uint h, int nbSphere,float dt, Sphere* ptrDevTabSphere);

extern __host__ void uploadGPU(Sphere* tabValue);
/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
RayTracing::RayTracing(const Grid& grid, uint w, uint h, int nbSphere, float dt/*, Sphere* ptrTabSphere*/) :
	Animable_I<uchar4>(grid, w, h, "RayTracig_CUDA_RGBA_uchar4"), variateurAnimation(Interval<float>(0.0, 1.0), dt)
    {

    this->nbSphere = nbSphere;
//    this->ptrTabSphere = new Sphere[nbSphere];
    this->wlocal = w;
    this->hlocal = h;
    this->dt = dt; // protected dans Animable

    SphereCreator sphereCreator(nbSphere, w, h);
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    // toGM(ptrTabSphere);
    this->sizeOctet = nbSphere * sizeof(Sphere);

    Device::malloc(&ptrDevTabSphere, sizeOctet);
    Device::memclear(ptrDevTabSphere, sizeOctet);

    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);

    //  toCM(ptrTabSphere);
    uploadGPU(ptrTabSphere);

    }
//TODO:
//void RayTracing::toGm(Sphere* ptrTabSphere)
//    {
//
//    // toGM(ptrTabSphere);
//    this->sizeOctet = nbSphere * sizeof(Sphere);
//
//    Device::malloc(&ptrDevTabSphere, sizeOctet);
//    Device::memclear(ptrDevTabSphere, sizeOctet);
//
//    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, sizeOctet);
//    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h/*, int nbSphere, Sphere* ptrTabSphere*/, const DomaineMath& domaineMath)
    {
//    Device::lastCudaError("fractale rgba uchar4 (before)"); // facultatif, for debug only, remove for release
    static int i = 2;

    if (i % 3 == 0)
	{
	rayTracingGM<<<dg,db>>>(ptrDevPixels, w, h, nbSphere, dt, ptrDevTabSphere);
	}
	else if (i % 3 == 1)
	{
	rayTracingCM<<<dg,db>>>(ptrDevPixels, w, h, dt);
	}
	else if (i % 3 == 2)
	{
	rayTracingSM<<<dg,db,sizeOctet>>>(ptrDevPixels, w, h, nbSphere, dt, ptrDevTabSphere);
	}
     //    Device::lastCudaError("fractale rgba uchar4 (after)"); // facultatif, for debug only, remove for release
//    i++;
}

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
{
//    t += dt;
dt = variateurAnimation.varierAndGet();
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

