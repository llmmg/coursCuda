#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"

#include <stdio.h>
#include "reductionADD.h"
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

//__global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);
__global__ void slice(int nbSlice, float* ptrDevResult);
__device__ void reduceIntraThread(float* tabSM, int nbSlice);

__device__ float fctPI(float x);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void slice(int nbSlice, float* ptrDevResult)
    {
    extern __shared__ float tabSM[];
    reduceIntraThread(tabSM, nbSlice);

    __syncthreads();

    reductionADD<float>(tabSM, ptrDevResult);

    }
/**
 * output : void required !!
 */

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void reduceIntraThread(float* tabSM, int nbSlice)
    {
    //int sumThread = 0;
//    int nbrSlice= ??
    const int NB_THREAD = Indice1D::nbThread();
    const int TID_LOCAL = Indice1D::tidLocal();
//    const int NB_THREAD_LOCAL = Indice1D::nbThreadLocal();
    const int TID = Indice1D::tid();

    const float DX = 1.0 / (float) nbSlice;

//    int s = TID_LOCAL;
    int s = TID;
    float derp = 0;

    while (s < nbSlice)
	{
//	sumThread += aire (,);
	derp += fctPI(s * DX);

	s += NB_THREAD;
	}

    tabSM[TID_LOCAL] = derp;
//    tabSM[TID_LOCAL] = TID ; //n*(n-1)/2=>523'776
    //tabSM[TID_LOCAL]=1; => 1024
    }

__device__ float fctPI(float x)
    {
    return 4.0f / (1.0f + (x * x));
    }
//
//__device__ float aire()
//    {
//    return
//    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

