#include "hip/hip_runtime.h"
#include "Slice.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void slice(int nbSlice, float* ptrDevResult);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice::Slice(const Grid& grid, int nbSlice)
    {

    this->sizeOctetSM = sizeof(float) * grid.db.x * grid.db.y;
    this->sizeOctetGM = sizeof(float);

    // MM
	{
	    {
	    //malloc modifie le contenu du ptr

	    Device::malloc(&ptrDevGMResult, sizeOctetGM);
	    Device::memclear(ptrDevGMResult, sizeOctetGM);

	    }
	Device::lastCudaError("bllblbll"); // temp debug, facultatif
	}

    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
    }
//
Slice::~Slice(void)
    {
    //MM (device free)
	{
	Device::free(ptrDevGMResult);
//	Device::free(ptrDevV2);
//	Device::free(ptrDevW);

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice::run()
    {
    Device::lastCudaError("addVecteur (before)"); // temp debug
    slice<<<dg,db,sizeOctetSM>>>(nbSlice,ptrDevGMResult); // assynchrone
    Device::lastCudaError("addVecteur (after)"); // temp debug

//    Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM(Device->Host)
	{
	Device::memcpyDToH(&result, ptrDevGMResult, sizeOctetGM); // barriere synchronisation implicite

	}

    result /= (float) nbSlice;
    }

float Slice::getResult()
    {
    return this->result;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

