#include "hip/hip_runtime.h"
#include <iostream>

#include "Device.h"
#include "Histogramme.h"
#include "AleaTools.h"
#include "DataCreator.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void histogramme(int* tabDevInput, int* tabDevHisto, int nbData,int dataMax);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

/**
 * tabData[i] in [0,255] */

Histogramme::Histogramme(const Grid& grid, int* ptrtabFrequence, int dataMax)
    {

    this->dg = grid.dg;
    this->db = grid.db;
    this->dataMax = dataMax;
    this->ptrTabFrequence = ptrtabFrequence;

    // init(tabFrequence, 256, 0);

    //Data
	{
	DataCreator dataCreator(dataMax);

	this->tabData = dataCreator.getTabData();
	this->nbData = dataCreator.getLength();

	this->sizeOctetHisto = sizeof(int) * dataMax;
	this->sizeOctetData = sizeof(int) * this->nbData;
	}

    //mem management
	{
	Device::malloc(&ptrTabIn, sizeOctetData);
	Device::memclear(ptrTabIn, sizeOctetData);
	Device::memcpyHToD(ptrTabIn, tabData, sizeOctetData);

	Device::malloc(&ptrTabOut, sizeOctetHisto);
	Device::memclear(ptrTabOut, sizeOctetHisto);
	}

    }

Histogramme::~Histogramme(void)
    {
//    delete[] tabData;
//    delete[] ptrTabFrequence;

    Device::free(ptrTabIn);
    Device::free(ptrTabOut);

    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Histogramme::run()
    {
    Device::lastCudaError("Histogramme (before)"); // temp debug
//    histogramme<<<dg, db, sizeOctetHisto>>>(tabData, tabFrequence, n);
    histogramme<<<dg, db, sizeOctetHisto>>>(ptrTabIn, ptrTabOut,nbData,dataMax);
    Device::lastCudaError("Histogramme (after)"); // temp debug

    // MM (Device -> Host)
	{
	//HANDLE_ERROR(hipMemcpy(tabHisto, tabDevHisto, sizeOctetHisto, hipMemcpyDeviceToHost)); // barriere synchronisation implicite
	}
    Device::synchronize();
    Device::memcpyDToH(ptrTabFrequence, ptrTabOut, sizeOctetHisto);
    }

int* Histogramme::getHistogramme()
    {
    return ptrTabFrequence;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
