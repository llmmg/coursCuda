#include "hip/hip_runtime.h"
#include "Indice1D.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "MathTools.h"
#include "reductionADD.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void histogramme(int* tabDevInput, int* tabDevHisto, int n, int nbData);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void initSM(int* tabSM, int n);
__device__ void reduceIntraThread(int* tabDevInput, int * tabSM, int nbData);
__device__ void reduceInterBlock(int* tabSM, int* tabDevHisto, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void histogramme(int* tabDevInput, int* tabDevHisto, int nbData,int dataMax)
    {
    extern __shared__ int tabSM[];

    //255=dataMax
    initSM(tabSM, dataMax);

    reduceIntraThread(tabDevInput, tabSM, nbData);
    __syncthreads();

    reduceInterBlock(tabSM, tabDevHisto, dataMax);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void initSM(int* tabSM, int n)
    {

    const int TID_LOCAL=Indice1D::tidLocal();
    const int NB_THREAD_LOCAL= Indice1D::nbThreadBlock();

    int s = TID_LOCAL;
    while (s < n)
	{
	tabSM[s] = 0;
	s += NB_THREAD_LOCAL;
	}
    }

__device__ void reduceIntraThread(int* tabDevInput, int* tabSM, int nbData)
    {
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    int s = TID;

    while (s < nbData)
	{
	int size = tabDevInput[s];
	atomicAdd(&tabSM[size], 1);
	s += NB_THREAD;
	}
    }

__device__ void reduceInterBlock(int* tabSM, int* tabGM, int dataMax)
    {

    if (Indice2D::tidLocal() == 0)
    	{
    	for (int i = 0; i < dataMax; i++)
    	    {
    	    atomicAdd(&tabGM[i], tabSM[i]);
    	    }
    	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
