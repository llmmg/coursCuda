#include "hip/hip_runtime.h"
#include "MonteCarloMulti.h"

#include "Device.h"
#include "MonteCarlo.h"

#include "Chrono.h"
#include "cudaTools.h"

#include <iostream>
using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
//extern __global__ void monteCarloMulti(hiprandState* tabDevGeneratorGM, int nbFlechettes, float m,int* ptrDevNx);
//extern __global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
MonteCarloMulti::MonteCarloMulti(const Grid& grid, int nbFlechettes, float m)
    {
    this->grid = grid;

    this->nbFlechette = nbFlechettes;
    this->m = m;

    this->nbDevice = Device::getDeviceCount();
    this->tabResult = new int[nbDevice];

    }

MonteCarloMulti::~MonteCarloMulti(void)
    {
    delete[] tabResult;
    }

void MonteCarloMulti::process()
    {
    Chrono chrono;
//#pragma omp parallel for
    for (int deviceId = 0; deviceId < nbDevice; deviceId++)
	{
	Device::setDevice(deviceId);

	MonteCarlo montecarlo(grid, nbFlechette / nbDevice, m);

	montecarlo.process();
	//getCountFlechettesSousCourbe dans MonteCarlo (pas la multi)
	int r = montecarlo.getCountFlechettesSousCourbe();

	tabResult[deviceId] = r;

	Device::printCurrent();
	}
    reduce(tabResult);

    result = (float) nbFlechTot / (float) nbFlechette * m;

    chrono.stop();
    cout << "ElapseTime: " << chrono.getElapseTimeS() << " (s)" << endl;
    //para: 0.149814s
    //seq: 0.42455s
    }

float MonteCarloMulti::getResult()
    {
    return this->result;
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
void MonteCarloMulti::reduce(int* tab)
    {
    this->nbFlechTot = 0;
    for (int i = 0; i < nbDevice; i++)
	{
	nbFlechTot += tab[i];
	}

    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

