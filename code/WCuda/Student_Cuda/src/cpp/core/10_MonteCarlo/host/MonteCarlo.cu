#include "hip/hip_runtime.h"
#include "Device.h"
#include "MonteCarlo.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
extern __global__ void monteCarlo(hiprandState* tabDevGeneratorGM, int nbFlechettes, float m,int* ptrDevNx);
extern __global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
MonteCarlo::MonteCarlo(const Grid& grid, int nbFlechettes, float m)
    {
    this->dg = grid.dg;
    this->db = grid.db;

    this->nbFlechette = nbFlechettes;
    this->m = m;
    this->sizeOctetSM = grid.db.x * grid.db.y * grid.db.z* sizeof(int);
    this->sizeOctetGM = sizeof(int);

    int nbThreads = grid.threadCounts();
    size_t sizeGen = sizeof(hiprandState) * nbThreads;

    Device::malloc(&ptrDevNx, sizeOctetGM);
    Device::memclear(ptrDevNx, sizeOctetGM);

    Device::malloc(&ptrDevGenerator, sizeGen);
    Device::memclear(ptrDevGenerator, sizeGen);

    setup_kernel_rand<<<dg,db>>>(ptrDevGenerator, Device::getDeviceId());

}

MonteCarlo::~MonteCarlo(void)
{
//MM (device free)
    {
    Device::free(ptrDevNx);
    Device::free(ptrDevGenerator);
    }
}

void MonteCarlo::process()
{
    monteCarlo<<<dg,db,sizeOctetSM>>>(ptrDevGenerator,nbFlechette, m,ptrDevNx);

    Device::memcpyDToH(&nbThrowRes, ptrDevNx, sizeOctetGM);

    result = (float)nbThrowRes / (float)nbFlechette *m;
}

float MonteCarlo::getResult()
{
return this->result;
}
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

