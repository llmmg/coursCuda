#include "hip/hip_runtime.h"
#include "Device.h"
#include "MonteCarlo.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
extern __global__ void monteCarlo(hiprandState* tabDevGeneratorGM, int nbFlechettes, float m,int* ptrDevNx);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
MonteCarlo::MonteCarlo(const Grid& grid, int nbFlechettes, float m)
    {
    this->dg = grid.dg;
    this->db = grid.db;

    this->nbFlechette = nbFlechettes;
    this->m = m;
    this->sizeOctetSM = grid.db.x * grid.db.y * sizeof(int);
    this->sizeOctetGM = sizeof(int);

    int nbThreads = grid.threadCounts();
    size_t sizeGen = sizeof(hiprandState) * nbThreads;

    Device::malloc(&ptrDevNx, sizeOctetGM);
    Device::memclear(ptrDevNx, sizeOctetGM);

    Device::malloc(&ptrDevGenerator, sizeGen);
    Device::memclear(ptrDevGenerator, sizeGen);

    }

MonteCarlo::~MonteCarlo(void)
    {
//MM (device free)
	{
	Device::free(ptrDevNx);
	}
    }

void MonteCarlo::process()
    {
    monteCarlo<<<dg,db,sizeOctetSM>>>(ptrDevGenerator,nbFlechette, m,ptrDevNx);

//    Device::memcpyDToH(&result, ptrDevNx, sizeOctetGM);
    }

float MonteCarlo::getResult()
    {
    return this->result;
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

