#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include "reductionADD.h"

#include <hiprand/hiprand_kernel.h>
#include <limits.h>
#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void monteCarlo(hiprandState* tabDevGeneratorGM, int nbFlechettes, float m);
__global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId);
static __device__ float f(float x);
__device__ void reduceIntraThread(int* tab_SM, int nbFlechette, hiprandState* tabDevGeneratorGM, float m);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * output : void required !!
 */
__global__ void monteCarlo(hiprandState* tabDevGeneratorGM, int nbFlechettes, float m, int* ptrDevNx)
    {
    extern __shared__ int tab_SM[];

    reduceIntraThread(tab_SM, nbFlechettes, tabDevGeneratorGM, m);
    __syncthreads();

    //Reduce
    //=>global mem
    reductionADD<int>(tab_SM, ptrDevNx);
    }

__device__ void reduceIntraThread(int* tab_SM, int nbFlechette, hiprandState* tabDevGeneratorGM, float m)
    {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();
    const int TID_LOCAL = threadIdx.x;
    // Global Memory -> Register (optimization)

    hiprandState localGenerator = tabDevGeneratorGM[TID];
    float xAlea;
    float yAlea;
    float y;
    int nx = 0;
    for (int i = 1; i <= nbFlechette / NB_THREAD; i++)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator) * m;

	y = f(xAlea);
	if (y >= yAlea)
	    {
	    nx++;
	    }

	}

    //Register -> Global Memory
    //Necessaire si on veut utiliser notre generator
    // - dans dautre kernel
    // - avec dautres nombres aleatoires !

    tabDevGeneratorGM[TID] = localGenerator;

    tab_SM[TID_LOCAL] = nx;
    }

__device__ float f(float x)
    {
    return 4.0 / (1.0 + (x * x));
    }

__global__ void setup_kernel_rand(hiprandState* tabDevGenerator, int deviceId)
    { // Customisation du generator:
// Proposition, au lecteur de faire mieux !
// Contrainte : Doit etre différent d'un GPU à l'autre
// Contrainte : Doit etre différent dun thread à lautre
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;
    hiprand_init(seed, sequenceNumber, offset, &tabDevGenerator[TID]);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

